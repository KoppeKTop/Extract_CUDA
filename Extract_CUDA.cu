#include "hip/hip_runtime.h"
#define _DEBUG
#undef _DEBUG

#define _SPEED_TEST
#undef _SPEED_TEST

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <cutil.h>
#include <multithreading.h>
#include "Extract_CUDA.h"
#include "cuda_helper.h"
#include "counting.cuh"


///////////////////////////////////////////////////////////////////////////////
// Common host and device function 
///////////////////////////////////////////////////////////////////////////////
//ceil(a / b)
extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//floor(a / b)
extern "C" int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
extern "C" int iAlignDown(int a, int b){
    return a - a % b;
}


extern "C" int getCube(char * cube, unsigned int &drug, t_params * params);
extern "C" int getParams(t_params * params, char * filename);
extern "C" void dump_cube(char *cube, int dim, char *fname);
extern "C" int load_dump(char *cube, int dim, char *fname);

// PThreads syncronization (sync_pthreads.cpp)
extern "C" void sync_pthreads();
extern "C" void init_sync(int N);

///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////
const int    PATH_N = 2*264*264*264;
const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

int strlen(char * s)
{
    int N;
    for (N = 0; *s != '\0'; s++)
        N++;
    return N;
}

void reverse(char * s)
{
    int c, i, j;
    for (i = 0, j = strlen(s)-1; i < j; i++, j--)
    {
        c = s[i];
        s[i]=s[j];
        s[j] = c;
    }
}

char * itoa(int N)
{
    int i, sign;
    char * s;
    s = (char *)malloc(10 * sizeof(char));
    if ((sign = N) < 0)
        N = -N;
    i = 0;
    do
    {
        s[i++] = N % 10 + '0';
    } while((N /= 10) >0);
    if (sign < 0) s[i++] = '-';
    s[i] = '\0';
    reverse(s);
    return s;
}

void myPrint(char * msg, char * fname)
{
    FILE * fp = fopen(fname, "a");
    if (fp)
    {
        fprintf(fp, msg);
        fprintf(fp, "\n");
        fclose(fp);
    }
}

///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    // Plan:
    // load cube from file
    // send it to gpu
    // generate initial randoms
    // 
    float
        *d_Rand;

    double
        gpuTime;

    int i,j;
    unsigned int hTimer, hSumTimer;
    char ** iniFilename;
    iniFilename = new (char *);
    iniFilename[0] = new char[256];

    t_params params;
    cutGetCmdLineArgumentstr( argc, (const char **)argv, "ini", iniFilename);
    if (iniFilename[0] == NULL) return 1;

    if (getParams(&params, iniFilename[0]) != 0)
    {
        printf("Oooops!\n");
        return 1;
    }

    char *cube;//, *cube_fin;
    cube = 0;
    char *cube_gpu;
    char *s;

    unsigned int *part_cnt;
    unsigned int *part_cnt_gpu, *z_gpu;
    unsigned int *sum_gpu;
    unsigned int drug = 0;
    dim3 vol_dim = params.vol_dim;
    // const int cells = params.vol_dim;
    const int all_cells = vol_dim.x * vol_dim.y * vol_dim.z;
    const int thickness = params.thickness;


    dim3 dimBlock(8, 8, 8);
    dim3 dimBlock_clear(8, 8);
    int grid_dim_x = vol_dim.x/16 + ( vol_dim.x % 16 == 0 )?0:1;
    int grid_dim_y = vol_dim.y/16 + ( vol_dim.y % 16 == 0 )?0:1;
    int grid_dim_z = vol_dim.z/16 + ( vol_dim.z % 16 == 0 )?0:1;
    dim3 dimGrid(grid_dim_x, grid_dim_y);

    CUT_DEVICE_INIT(argc, argv);
    CUT_SAFE_CALL( cutCreateTimer(&hTimer) );
    CUT_SAFE_CALL( cutCreateTimer(&hSumTimer) );

    printf("Initializing data\n", PATH_N);
    part_cnt = (unsigned int *)malloc(sizeof(unsigned int));
    cube     = (char *)malloc(all_cells * sizeof(char));

    CUDA_SAFE_CALL( hipMalloc((void **)&cube_gpu, all_cells * sizeof(char)) );
    CUDA_SAFE_CALL( hipMalloc((void **)&d_Rand, RAND_N * sizeof(float)) );
    CUDA_SAFE_CALL( hipMalloc((void **)&part_cnt_gpu, sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc((void **)&sum_gpu, 512 * sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc((void **)&z_gpu, sizeof(int)) );
#ifdef _DEBUG
    unsigned int *debug_gpu, *dbg_cpu;
    unsigned int dbg_sum;
    CUDA_SAFE_CALL( hipMalloc((void **)&debug_gpu, 512*sizeof(int)) );
    dbg_cpu = (unsigned int *)malloc(512*sizeof(unsigned int));
    for (i = 0; i < 512; i++) dbg_cpu[i] = 0;
#endif

    if (params.dump_from)
    {
        if (load_dump(cube, vol_dim, params.dump_from) == 0) return 1;
    }
    else 
        if (getCube(cube, drug, &params) < 0) return 1;
        else dump_cube(cube, vol_dim, params.dump_to);

    CUDA_SAFE_CALL( hipMemcpy(cube_gpu, cube, all_cells * sizeof(char), hipMemcpyHostToDevice));
    
    printf("Loading CPU and GPU twisters configurations...\n");
    srand((unsigned int) time(NULL));

    myPrint("Let's try some iterations...\n", params.print_to);
    myPrint((s=itoa((drug))), params.print_to);
    free(s);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutResetTimer(hSumTimer) );
    CUT_SAFE_CALL( cutStartTimer(hSumTimer) );

    for (i=1; i <= params.max_iter; i++)
    {
    #ifdef _DEBUG
        CUDA_SAFE_CALL( hipMemcpy(debug_gpu, dbg_cpu, sizeof(int), hipMemcpyHostToDevice));
    #endif

    #ifdef _SPEED_TEST
        CUT_SAFE_CALL( cutStartTimer(hTimer) );
    #endif

        for (j = 0; j < grid_dim; j++)
        {
        #ifdef _DEBUG
            ca_step<<<dimGrid, dimBlock>>>(cube_gpu, 0, d_Rand, j, debug_gpu);
        #else
            ca_step<<<dimGrid, dimBlock>>>(cube_gpu, 0, d_Rand, j);
        #endif
            CUT_CHECK_ERROR("Even step execution failed\n");
            CUDA_SAFE_CALL( hipDeviceSynchronize() );
        }

    #ifdef _SPEED_TEST
        cutStopTimer(hTimer);
        printf("Even step time: %f ms\n\n", cutGetTimerValue(hTimer));
        cutResetTimer(hTimer);
    #endif

    #ifdef _DEBUG
        CUDA_SAFE_CALL( hipMemcpy(dbg_cpu, debug_gpu, sizeof(int), hipMemcpyDeviceToHost));
        if (dbg_cpu[0] != 0)
        {
            printf("Borders crossed %i times!!!\n", dbg_cpu[0]);
            dbg_cpu[0] = 0;
        }
    #endif

    #ifdef _DEBUG
        CUDA_SAFE_CALL( hipMemcpy(debug_gpu, dbg_cpu, sizeof(int), hipMemcpyHostToDevice));
    #endif

    #ifdef _SPEED_TEST
        CUT_SAFE_CALL( cutStartTimer(hTimer) );
    #endif

        for (j = 0; j < grid_dim; j++)
        {
    #ifdef _DEBUG
            ca_step<<<dimGrid, dimBlock>>>(cube_gpu, 1, d_Rand, j, debug_gpu);
    #else
            ca_step<<<dimGrid, dimBlock>>>(cube_gpu, 1, d_Rand, z_gpu);
    #endif
            CUT_CHECK_ERROR("Odd step execution failed\n");
            CUDA_SAFE_CALL( hipDeviceSynchronize() );
        }

    #ifdef _SPEED_TEST
        cutStopTimer(hTimer);
        printf("Odd step time: %f ms\n\n", cutGetTimerValue(hTimer));
        cutResetTimer(hTimer);
    #endif


    #ifdef _DEBUG
        CUDA_SAFE_CALL( hipMemcpy(dbg_cpu, debug_gpu, sizeof(int), hipMemcpyDeviceToHost));
        if (dbg_cpu[0] != 0)
        {
            printf("Borders crossed %i times!!!\n", dbg_cpu[0]);
            dbg_cpu[0] = 0;
        }
    #endif

    #ifdef _SPEED_TEST
        CUT_SAFE_CALL( cutStartTimer(hTimer) );
    #endif
            
        //for (j = 0; j < grid_dim; j++)
            //{
                //CUDA_SAFE_CALL( hipMemcpy(z_gpu, &j, sizeof(int), hipMemcpyHostToDevice));
        clear_cells<<<dimGrid, dimBlock_clear>>>(cube_gpu);
        CUT_CHECK_ERROR("Clearing failed\n");
        CUDA_SAFE_CALL( hipDeviceSynchronize() );                
        //}

    #ifdef _SPEED_TEST
        cutStopTimer(hTimer);
        printf("Clar cells time: %f ms\n\n", cutGetTimerValue(hTimer));
        cutResetTimer(hTimer);
    #endif


        if (i % params.dump_every == 0)
        {
            CUDA_SAFE_CALL( hipMemcpy(cube, cube_gpu, all_cells * sizeof(char), hipMemcpyDeviceToHost));
            dump_cube(cube, cells, params.dump_to);
        }
        printf("%i-th iteration end\n", i);
    }
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hSumTimer) );
    gpuTime = cutGetTimerValue(hSumTimer);
    printf("Iteration time  : %f \n", (double)gpuTime/i);

    myPrint("exiting", params.print_to);
    printf("Shutting down...\n");
    CUDA_SAFE_CALL( hipFree(d_Rand) );
    CUDA_SAFE_CALL( hipFree(part_cnt_gpu) );
    CUDA_SAFE_CALL( hipFree(cube_gpu) );
    CUDA_SAFE_CALL( hipFree(sum_gpu) );
    CUDA_SAFE_CALL( hipFree(z_gpu) );
#ifdef _DEBUG
    CUDA_SAFE_CALL( hipFree(debug_gpu) );
    free(dbg_cpu);
#endif
    free(part_cnt);
    free(cube);

    CUT_SAFE_CALL( cutDeleteTimer( hTimer) );

    CUT_EXIT(argc, argv);
}

